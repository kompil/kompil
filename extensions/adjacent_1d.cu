#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <THC/THCAtomics.cuh>

#include <vector>

#define MAX_THREADS 256
#define BATCH_PER_ITER 16
#define BATCH_PER_ITER_FORWARD BATCH_PER_ITER
#define BATCH_PER_ITER_BACKWARD BATCH_PER_ITER

namespace kernels::adjacent_1d {

__device__ inline void get_item(long idx, int s, int* c, int* x)
{
    *x = idx % s;
    *c = idx / s;
}

template <typename T>
__device__ inline T round_div_int(T val, T div)
{
    T output = (val + div / 2) / div;
    return output;
}

template <typename scalar_t>
__global__ void forward(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> batch_in, // {N, in_c, in_s}
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> weights, // {in_c, ker, out_c, out_s}
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> bias, // {out_c, out_s}
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output, // {N, out_c, out_s}
    const int batch_size,
    const int in_c,
    const int in_s,
    const int ker,
    const int out_c,
    const int out_s)
{
    const long out_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (out_idx >= out_c * out_s) {
        return;
    }

    int out_ch, out_x;
    get_item(out_idx, out_s, &out_ch, &out_x);

    const int in_x = round_div_int<long>(static_cast<long>(out_x) * (in_s - ker), out_s - 1);
    const int max_x = std::min(ker, in_s - in_x);

    const scalar_t curr_bias = bias[out_ch][out_x];

    scalar_t b_curr_acc[BATCH_PER_ITER_FORWARD];
    scalar_t b_input[BATCH_PER_ITER_FORWARD];

    for (int batch = 0; batch < batch_size; batch += BATCH_PER_ITER_FORWARD) {

        int id_for_switch = std::min(batch_size - batch, BATCH_PER_ITER_FORWARD);

        #pragma unroll
        for (int batch2 = 0; batch2 < BATCH_PER_ITER_FORWARD; ++batch2) {
            if (batch2 < id_for_switch) {
                b_curr_acc[batch2] = curr_bias;
            }
        }

        for (int in_ch = 0; in_ch < in_c; ++in_ch) {

            for (int ker_x = 0; ker_x < max_x; ++ker_x) {
                const int curr_in_x = in_x + ker_x;

                const scalar_t weight = weights[in_ch][ker_x][out_ch][out_x];

                #pragma unroll
                for (int batch2 = 0; batch2 < BATCH_PER_ITER_FORWARD; ++batch2) {
                    if (batch2 < id_for_switch) {
                        b_input[batch2] = batch_in[batch + batch2][in_ch][curr_in_x];
                    }
                }

                #pragma unroll
                for (int batch2 = 0; batch2 < BATCH_PER_ITER_FORWARD; ++batch2) {
                    if (batch2 < id_for_switch) {
                        b_curr_acc[batch2] += weight * b_input[batch2];
                    }
                }
            }
        }

        #pragma unroll
        for (int batch2 = 0; batch2 < BATCH_PER_ITER_FORWARD; ++batch2) {
            if (batch2 < id_for_switch) {
                output[batch + batch2][out_ch][out_x] = b_curr_acc[batch2];
            }
        }
    }
}

template <typename scalar_t>
__global__ void backward(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_input, // {N, in_c, in_s}
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> grad_weights, // {in_c, ker, out_c, out_s}
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> batch_in, // {N, in_c, in_s}
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> weights, // {in_c, ker, out_c, out_s}
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_output, // {N, out_c, out_s}
    const int batch_size,
    const int in_c,
    const int in_s,
    const int ker,
    const int out_c,
    const int out_s)
{
    const long in_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (in_idx >= in_c * in_s) {
        return;
    }

    int in_ch, in_x;
    get_item(in_idx, in_s, &in_ch, &in_x);

    int start_out_x;
    int end_out_x;

    if (in_s == ker) {
        start_out_x = 0;
        end_out_x = out_s;
    } else {
        start_out_x = round_div_int<long>(static_cast<long>(in_x - ker - 1) * (out_s - 1), in_s - ker);
        end_out_x = round_div_int<long>(static_cast<long>(in_x + ker + 1) * (out_s - 1), in_s - ker);
        start_out_x = std::max<int>(0, start_out_x);
        end_out_x = std::min<int>(out_s, end_out_x);
    }

    scalar_t b_input[BATCH_PER_ITER_BACKWARD];
    scalar_t b_grad_input[BATCH_PER_ITER_BACKWARD];
    scalar_t b_grad_output[BATCH_PER_ITER_BACKWARD];

    for (int batch = 0; batch < batch_size; batch += BATCH_PER_ITER_BACKWARD) {

        int id_for_switch = std::min(batch_size - batch, BATCH_PER_ITER_BACKWARD);

        #pragma unroll
        for (int batch2 = 0; batch2 < BATCH_PER_ITER_BACKWARD; ++batch2) {
            if (batch2 < id_for_switch) {
                b_input[batch2] = batch_in[batch + batch2][in_ch][in_x];
                b_grad_input[batch2] = 0;
            }
        }

        for (int out_x = start_out_x; out_x < end_out_x; ++out_x) {
            // Extract which ker_x this output pixel refers to (TODO verify sign)
            const int ker_x = in_x - round_div_int<long>(static_cast<long>(out_x) * (in_s - ker), out_s - 1);
            // Pass if the pixel does not match the one for this cuda kernel
            if (ker_x < 0 || ker_x >= ker) {
                continue;
            }

            for (int out_ch = 0; out_ch < out_c; ++out_ch) {

                scalar_t* grad_weight_ptr = &grad_weights[in_ch][ker_x][out_ch][out_x];

                // Access required data
                const scalar_t weight = weights[in_ch][ker_x][out_ch][out_x];
                const scalar_t grad_weight = *grad_weight_ptr;

                #pragma unroll
                for (int batch2 = 0; batch2 < BATCH_PER_ITER_BACKWARD; ++batch2) {
                    if (batch2 < id_for_switch) {
                        b_grad_output[batch2] = grad_output[batch + batch2][out_ch][out_x];
                    }
                }

                // Grad weight and Grad input add
                scalar_t grad_weight_add{grad_weight};

                #pragma unroll
                for (int batch2 = 0; batch2 < BATCH_PER_ITER_BACKWARD; ++batch2) {
                    if (batch2 < id_for_switch) {
                        scalar_t b2_grad_output = b_grad_output[batch2];
                        b_grad_input[batch2] += weight * b2_grad_output;
                        grad_weight_add += b_input[batch2] * b2_grad_output;
                    }
                }

                *grad_weight_ptr = grad_weight_add;
            }
        }

        // Grad input
        #pragma unroll
        for (int batch2 = 0; batch2 < BATCH_PER_ITER_BACKWARD; ++batch2) {
            if (batch2 < id_for_switch) {
                grad_input[batch + batch2][in_ch][in_x] = b_grad_input[batch2];
            }
        }
    }
}

} // namespace kernels::adjacent_1d

torch::Tensor _cuda_adjacent_1d_forward(
    torch::Tensor batch_in, // tensor of shape (N, in_c, in_s)
    int out_c,
    int out_s,
    int ker,
    torch::Tensor weights, // tensor of shape (in_c, ker, out_c, out_s)
    torch::Tensor bias) // tensor of shape (out_c, out_s)
{
    // Get values
    const int batch_size = batch_in.size(0);
    const int in_c = batch_in.size(1);
    const int in_s = batch_in.size(2);
    const long out_items = out_c * out_s;
    // Define threads & blocks
    const int threads = MAX_THREADS;
    const dim3 blocks(out_items / threads + (out_items % threads != 0));
    // Prepare data
    auto output = torch::empty(at::IntArrayRef({batch_size, out_c, out_s}), batch_in.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(batch_in.scalar_type(), "_cuda_kernel_adjacent_1d_forward", ([&] {
        using namespace kernels::adjacent_1d;

        forward<scalar_t><<<blocks, threads>>>(
            batch_in.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            weights.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            bias.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            batch_size,
            in_c,
            in_s,
            ker,
            out_c,
            out_s);
    }));

    // Check for errors
    auto error = hipGetLastError();
    if (error) {
        throw std::runtime_error{"CUDA kernel error: " + std::string{hipGetErrorString(error)}};
    }

    return output;
}

std::vector<torch::Tensor> _cuda_adjacent_1d_backward(
    torch::Tensor batch_in, // tensor of shape (N, in_c, in_s)
    int out_c,
    int out_s,
    int ker,
    torch::Tensor weights, // tensor of shape (in_c, ker, out_c, out_s)
    torch::Tensor bias, // tensor of shape (out_c, out_s)
    torch::Tensor grad_output) // tensor of shape (N, out_c, out_s)
{
    // Get values
    const int batch_size = batch_in.size(0);
    const int in_c = batch_in.size(1);
    const int in_s = batch_in.size(2);
    const long in_items = in_c * in_s;
    // Define threads & blocks
    const int threads = MAX_THREADS;
    const dim3 blocks(in_items / threads + (in_items % threads != 0));
    // Prepare data
    auto grad_input = torch::zeros_like(batch_in, batch_in.options());
    auto grad_weights = torch::zeros_like(weights, batch_in.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(batch_in.scalar_type(), "_cuda_kernel_adjacent_1d_backward", ([&] {
        using namespace kernels::adjacent_1d;

        backward<scalar_t><<<blocks, threads>>>(
            grad_input.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            grad_weights.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            batch_in.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            weights.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            batch_size,
            in_c,
            in_s,
            ker,
            out_c,
            out_s);
    }));

    // Check for errors
    auto error = hipGetLastError();
    if (error) {
        throw std::runtime_error{"CUDA kernel error: " + std::string{hipGetErrorString(error)}};
    }

    // Calculate bias which is the sum of the output grads along batches
    auto grad_bias = torch::sum(grad_output, 0);

    return {grad_input, grad_weights, grad_bias};
}

